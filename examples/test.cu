#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

//#define CHUNK_SIZE (2ULL * 1024 * 1024)
#define CHUNK_SIZE 4ULL

#define WAIT_TIME   10000000000L // about 5 seconds on RTX3080

__global__ void 
loop(volatile uint64_t *page, uint64_t x)
{
  uint64_t y = x;
  volatile uint64_t *ptr;
  uint64_t clk0;
  uint64_t clk1;
  
  while (y == x) {
    for (ptr = (uint64_t *)page[0]; ptr != page; ptr = (uint64_t *)ptr[0])
      ++ptr[2];
    
    clk0 = clock64();
    clk1 = 0;
    while (clk1 < WAIT_TIME)
      clk1 = clock64() - clk0;
    
    y = ptr[1];
  }
}

__global__ void
put(uint8_t *page, unsigned page_index, uint64_t value)
{
  page[page_index * 4096] = value;
}

void
access_page(uint8_t *page, unsigned page_index)
{
  put<<<1, 1>>>(page, page_index, 3);
  hipDeviceSynchronize();
  printf("page %u is accessed by gpu\n", page_index);
  fflush(stdout);
}

int 
main(int argc, char *argv[])
{
  uint8_t *chunk = NULL;
  uint8_t *chunk1 = NULL;
  
  hipDeviceReset();

  printf("hipDeviceReset is called\n");
  fflush(stdout);
  while (getchar() != '\n') {}
  
  // hoard a large address space
  hipMallocManaged(&chunk, CHUNK_SIZE);
  //cudaMalloc(&chunk, CHUNK_SIZE);
  printf("hipMallocManaged is called, (chunk) addr: %p, size: %llx\n", chunk, CHUNK_SIZE);

  hipMallocManaged(&chunk1, CHUNK_SIZE);
  //cudaMalloc(&chunk1, CHUNK_SIZE);
  printf("hipMallocManaged is called, (chunk1) addr: %p, size: %llx\n", chunk1, CHUNK_SIZE);
  fflush(stdout);
  while (getchar() != '\n') {}


  memset(chunk, 0, CHUNK_SIZE);
  printf("accessed by cpu\n");

  fflush(stdout);
  while (getchar() != '\n') {}

  access_page(chunk, 0);
  printf("accessed by gpu\n");

  fflush(stdout);
  while (getchar() != '\n') {}

  memset(chunk, 0, CHUNK_SIZE);
  printf("accessed by cpu again\n");

  fflush(stdout);
  while (getchar() != '\n') {}
  /*
  while (1) {
      printf("input page index\n");
      scanf("%u", &page_index);

      access_page(chunk, page_index);
  }
  */

  
  hipFree(chunk);
  hipFree(chunk1);
}


